﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>

#include "blocks.cuh"
#include "../utils.h"

void test_baseband() {

    const std::string signalFileAddress = "input.bin";
    const int frameLen = 2 * 1024 * 1024;
    const float frequency = 0.1f;

    PartialFileReader fileReader;
    fileReader.setFileName(signalFileAddress);
    fileReader.openFile();
    int num_elements = fileReader.getTotalFileSizeInBytes() / sizeof(hipComplex);
    
    PartialFileWriter fileWriter;
    fileWriter.setFileName("output.bin");
    fileWriter.openFile();

    std::vector<hipComplex> h_inOut(frameLen);
    hipComplex* d_inOut;
    hipMalloc(&d_inOut, frameLen * sizeof(hipComplex));
    float* d_freq_init;
    hipMalloc(&d_freq_init, sizeof(float));

    int i{};
    while ((i + 1) * frameLen <= num_elements) {
        fileReader.readBinData(h_inOut, frameLen);
        hipMemcpyAsync(d_inOut, h_inOut.data(), frameLen * sizeof(hipComplex), hipMemcpyHostToDevice);

        Baseband << <12, 1024 >> > (d_inOut, d_freq_init, frameLen, frequency);
        Baseband_Update_State<<<1, 1>>>(d_freq_init, frequency, frameLen);

        hipMemcpy(h_inOut.data(), d_inOut, frameLen * sizeof(hipComplex), hipMemcpyDeviceToHost);
        fileWriter.writeBinData(h_inOut, frameLen);
        i++;
    }

    fileReader.closeFile();
    fileWriter.closeFile();

    hipFree(d_inOut);
    hipFree(d_freq_init);
    gpuErrchk();
}

void test_bbfilter() {

    const std::string signalFileAddress = "input.bin";
    const std::string filterCoeffsFileAddress = "filter_coeffs.bin";
    const int frameLen = 2 * 1024 * 1024;

    PartialFileReader fileReader;
    fileReader.setFileName(signalFileAddress);
    fileReader.openFile();
    int num_elements = fileReader.getTotalFileSizeInBytes() / sizeof(hipComplex);

    PartialFileWriter fileWriter;
    fileWriter.setFileName("output.bin");
    fileWriter.openFile();

    std::vector<float> h_filterCoeffs;
    readBinData(h_filterCoeffs, filterCoeffsFileAddress);
    const int bbfilterLen = h_filterCoeffs.size();
    setBBFilterCoeffsConstMem(h_filterCoeffs.data(), h_filterCoeffs.size());

    std::vector<hipComplex> h_inOut(frameLen);
    hipComplex* d_input, *d_output;
    hipMalloc(&d_input, (frameLen + bbfilterLen - 1) * sizeof(hipComplex));
    hipMemsetAsync(d_input, 0.f, (bbfilterLen - 1) * sizeof(hipComplex));
    hipMalloc(&d_output, frameLen * sizeof(hipComplex));
    gpuErrchk();

    int i{};
    while ((i + 1) * frameLen <= num_elements) {
        fileReader.readBinData(h_inOut, frameLen);
        hipMemcpyAsync(d_input + bbfilterLen - 1, h_inOut.data(), frameLen * sizeof(hipComplex), hipMemcpyHostToDevice);

        BasebandFilter << <12, 1024 >> > (d_output, d_input, bbfilterLen, frameLen);
        hipMemcpyAsync(d_input, d_input + frameLen, (bbfilterLen - 1) * sizeof(hipComplex), hipMemcpyDeviceToDevice);

        hipMemcpy(h_inOut.data(), d_output, frameLen * sizeof(hipComplex), hipMemcpyDeviceToHost);
        fileWriter.writeBinData(h_inOut, frameLen);
        i++;
    }

    fileReader.closeFile();
    fileWriter.closeFile();

    hipFree(d_input);
    hipFree(d_output);
    gpuErrchk();
}

int main()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    //test_baseband();
    test_bbfilter();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
