﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "blocks.cuh"
#include "../utils.h"

void test_baseband() {

    const std::string signalFileAddress = "input.bin";
    const int frameLen = 2 * 1024 * 1024;
    const float frequency = 0.1f;

    PartialFileReader fileReader;
    fileReader.setFileName(signalFileAddress);
    fileReader.openFile();
    int num_elements = fileReader.getTotalFileSizeInBytes() / sizeof(hipComplex);
    
    PartialFileWriter fileWriter;
    fileWriter.setFileName("output.bin");
    fileWriter.openFile();

    std::vector<hipComplex> h_inOut(frameLen);
    hipComplex* d_inOut;
    hipMalloc(&d_inOut, frameLen * sizeof(hipComplex));
    float* d_freq_init;
    hipMalloc(&d_freq_init, sizeof(float));

    int i{};
    while ((i + 1) * frameLen <= num_elements) {
        fileReader.readBinData(h_inOut, frameLen);
        hipMemcpyAsync(d_inOut, h_inOut.data(), frameLen * sizeof(hipComplex), hipMemcpyHostToDevice);

        Baseband << <12, 1024 >> > (d_inOut, d_freq_init, frameLen, frequency);
        Baseband_Update_State<<<1, 1>>>(d_freq_init, frequency, frameLen);

        hipMemcpy(h_inOut.data(), d_inOut, frameLen * sizeof(hipComplex), hipMemcpyDeviceToHost);
        fileWriter.writeBinData(h_inOut, frameLen);
        i++;
    }

    fileReader.closeFile();
    fileWriter.closeFile();

    hipFree(d_inOut);
    hipFree(d_freq_init);
    gpuErrchk();
}

int main()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    test_baseband();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
